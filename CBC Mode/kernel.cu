#include "hip/hip_runtime.h"
/*
This version assigns one thread per 16 bytes of text.(one text block)
Stores the plaintext/ciphertext in registers.
Stores the encryption keys in shared memory.
Stores the S-boxes in shared memory.
The blocksize is 512.
*/

#include <iostream>
#include <fstream>
#include <sstream>
#include <chrono>

#include <hip/hip_runtime_api.h>
#include <>


typedef unsigned char uint8;

//Key generation constants
uint8 C1[] = { 0x51,0x7c,0xc1,0xb7,0x27,0x22,0x0a,0x94,0xfe,0x13,0xab,0xe8,0xfa,0x9a,0x6e,0xe0 };
uint8 C2[] = { 0x6d,0xb1,0x4a,0xcc,0x9e,0x21,0xc8,0x20,0xff,0x28,0xb1,0xd5,0xef,0x5d,0xe2,0xb0 };
uint8 C3[] = { 0xdb,0x92,0x37,0x1d,0x21,0x26,0xe9,0x70,0x03,0x24,0x97,0x75,0x04,0xe8,0xc9,0x0e };

//Encryption round keys
uint8 ek[272] = { 0 }; //272 bytes(17 round keys each 16 bytes)
					   //Decyription round keys
uint8 dk[272] = { 0 }; //272 bytes(17 round keys each 16 bytes)

					   //S-boxes
static const uint8 SB1[256] =
{
	0x63, 0x7C, 0x77, 0x7B, 0xF2, 0x6B, 0x6F, 0xC5, 0x30, 0x01, 0x67, 0x2B, 0xFE, 0xD7, 0xAB, 0x76,
	0xCA, 0x82, 0xC9, 0x7D, 0xFA, 0x59, 0x47, 0xF0, 0xAD, 0xD4, 0xA2, 0xAF, 0x9C, 0xA4, 0x72, 0xC0,
	0xB7, 0xFD, 0x93, 0x26, 0x36, 0x3F, 0xF7, 0xCC, 0x34, 0xA5, 0xE5, 0xF1, 0x71, 0xD8, 0x31, 0x15,
	0x04, 0xC7, 0x23, 0xC3, 0x18, 0x96, 0x05, 0x9A, 0x07, 0x12, 0x80, 0xE2, 0xEB, 0x27, 0xB2, 0x75,
	0x09, 0x83, 0x2C, 0x1A, 0x1B, 0x6E, 0x5A, 0xA0, 0x52, 0x3B, 0xD6, 0xB3, 0x29, 0xE3, 0x2F, 0x84,
	0x53, 0xD1, 0x00, 0xED, 0x20, 0xFC, 0xB1, 0x5B, 0x6A, 0xCB, 0xBE, 0x39, 0x4A, 0x4C, 0x58, 0xCF,
	0xD0, 0xEF, 0xAA, 0xFB, 0x43, 0x4D, 0x33, 0x85, 0x45, 0xF9, 0x02, 0x7F, 0x50, 0x3C, 0x9F, 0xA8,
	0x51, 0xA3, 0x40, 0x8F, 0x92, 0x9D, 0x38, 0xF5, 0xBC, 0xB6, 0xDA, 0x21, 0x10, 0xFF, 0xF3, 0xD2,
	0xCD, 0x0C, 0x13, 0xEC, 0x5F, 0x97, 0x44, 0x17, 0xC4, 0xA7, 0x7E, 0x3D, 0x64, 0x5D, 0x19, 0x73,
	0x60, 0x81, 0x4F, 0xDC, 0x22, 0x2A, 0x90, 0x88, 0x46, 0xEE, 0xB8, 0x14, 0xDE, 0x5E, 0x0B, 0xDB,
	0xE0, 0x32, 0x3A, 0x0A, 0x49, 0x06, 0x24, 0x5C, 0xC2, 0xD3, 0xAC, 0x62, 0x91, 0x95, 0xE4, 0x79,
	0xE7, 0xC8, 0x37, 0x6D, 0x8D, 0xD5, 0x4E, 0xA9, 0x6C, 0x56, 0xF4, 0xEA, 0x65, 0x7A, 0xAE, 0x08,
	0xBA, 0x78, 0x25, 0x2E, 0x1C, 0xA6, 0xB4, 0xC6, 0xE8, 0xDD, 0x74, 0x1F, 0x4B, 0xBD, 0x8B, 0x8A,
	0x70, 0x3E, 0xB5, 0x66, 0x48, 0x03, 0xF6, 0x0E, 0x61, 0x35, 0x57, 0xB9, 0x86, 0xC1, 0x1D, 0x9E,
	0xE1, 0xF8, 0x98, 0x11, 0x69, 0xD9, 0x8E, 0x94, 0x9B, 0x1E, 0x87, 0xE9, 0xCE, 0x55, 0x28, 0xDF,
	0x8C, 0xA1, 0x89, 0x0D, 0xBF, 0xE6, 0x42, 0x68, 0x41, 0x99, 0x2D, 0x0F, 0xB0, 0x54, 0xBB, 0x16
};

static const uint8 SB2[256] =
{
	0xE2, 0x4E, 0x54, 0xFC, 0x94, 0xC2, 0x4A, 0xCC, 0x62, 0x0D, 0x6A, 0x46, 0x3C, 0x4D, 0x8B, 0xD1,
	0x5E, 0xFA, 0x64, 0xCB, 0xB4, 0x97, 0xBE, 0x2B, 0xBC, 0x77, 0x2E, 0x03, 0xD3, 0x19, 0x59, 0xC1,
	0x1D, 0x06, 0x41, 0x6B, 0x55, 0xF0, 0x99, 0x69, 0xEA, 0x9C, 0x18, 0xAE, 0x63, 0xDF, 0xE7, 0xBB,
	0x00, 0x73, 0x66, 0xFB, 0x96, 0x4C, 0x85, 0xE4, 0x3A, 0x09, 0x45, 0xAA, 0x0F, 0xEE, 0x10, 0xEB,
	0x2D, 0x7F, 0xF4, 0x29, 0xAC, 0xCF, 0xAD, 0x91, 0x8D, 0x78, 0xC8, 0x95, 0xF9, 0x2F, 0xCE, 0xCD,
	0x08, 0x7A, 0x88, 0x38, 0x5C, 0x83, 0x2A, 0x28, 0x47, 0xDB, 0xB8, 0xC7, 0x93, 0xA4, 0x12, 0x53,
	0xFF, 0x87, 0x0E, 0x31, 0x36, 0x21, 0x58, 0x48, 0x01, 0x8E, 0x37, 0x74, 0x32, 0xCA, 0xE9, 0xB1,
	0xB7, 0xAB, 0x0C, 0xD7, 0xC4, 0x56, 0x42, 0x26, 0x07, 0x98, 0x60, 0xD9, 0xB6, 0xB9, 0x11, 0x40,
	0xEC, 0x20, 0x8C, 0xBD, 0xA0, 0xC9, 0x84, 0x04, 0x49, 0x23, 0xF1, 0x4F, 0x50, 0x1F, 0x13, 0xDC,
	0xD8, 0xC0, 0x9E, 0x57, 0xE3, 0xC3, 0x7B, 0x65, 0x3B, 0x02, 0x8F, 0x3E, 0xE8, 0x25, 0x92, 0xE5,
	0x15, 0xDD, 0xFD, 0x17, 0xA9, 0xBF, 0xD4, 0x9A, 0x7E, 0xC5, 0x39, 0x67, 0xFE, 0x76, 0x9D, 0x43,
	0xA7, 0xE1, 0xD0, 0xF5, 0x68, 0xF2, 0x1B, 0x34, 0x70, 0x05, 0xA3, 0x8A, 0xD5, 0x79, 0x86, 0xA8,
	0x30, 0xC6, 0x51, 0x4B, 0x1E, 0xA6, 0x27, 0xF6, 0x35, 0xD2, 0x6E, 0x24, 0x16, 0x82, 0x5F, 0xDA,
	0xE6, 0x75, 0xA2, 0xEF, 0x2C, 0xB2, 0x1C, 0x9F, 0x5D, 0x6F, 0x80, 0x0A, 0x72, 0x44, 0x9B, 0x6C,
	0x90, 0x0B, 0x5B, 0x33, 0x7D, 0x5A, 0x52, 0xF3, 0x61, 0xA1, 0xF7, 0xB0, 0xD6, 0x3F, 0x7C, 0x6D,
	0xED, 0x14, 0xE0, 0xA5, 0x3D, 0x22, 0xB3, 0xF8, 0x89, 0xDE, 0x71, 0x1A, 0xAF, 0xBA, 0xB5, 0x81
};

static const uint8 SB3[256] =
{
	0x52, 0x09, 0x6A, 0xD5, 0x30, 0x36, 0xA5, 0x38, 0xBF, 0x40, 0xA3, 0x9E, 0x81, 0xF3, 0xD7, 0xFB,
	0x7C, 0xE3, 0x39, 0x82, 0x9B, 0x2F, 0xFF, 0x87, 0x34, 0x8E, 0x43, 0x44, 0xC4, 0xDE, 0xE9, 0xCB,
	0x54, 0x7B, 0x94, 0x32, 0xA6, 0xC2, 0x23, 0x3D, 0xEE, 0x4C, 0x95, 0x0B, 0x42, 0xFA, 0xC3, 0x4E,
	0x08, 0x2E, 0xA1, 0x66, 0x28, 0xD9, 0x24, 0xB2, 0x76, 0x5B, 0xA2, 0x49, 0x6D, 0x8B, 0xD1, 0x25,
	0x72, 0xF8, 0xF6, 0x64, 0x86, 0x68, 0x98, 0x16, 0xD4, 0xA4, 0x5C, 0xCC, 0x5D, 0x65, 0xB6, 0x92,
	0x6C, 0x70, 0x48, 0x50, 0xFD, 0xED, 0xB9, 0xDA, 0x5E, 0x15, 0x46, 0x57, 0xA7, 0x8D, 0x9D, 0x84,
	0x90, 0xD8, 0xAB, 0x00, 0x8C, 0xBC, 0xD3, 0x0A, 0xF7, 0xE4, 0x58, 0x05, 0xB8, 0xB3, 0x45, 0x06,
	0xD0, 0x2C, 0x1E, 0x8F, 0xCA, 0x3F, 0x0F, 0x02, 0xC1, 0xAF, 0xBD, 0x03, 0x01, 0x13, 0x8A, 0x6B,
	0x3A, 0x91, 0x11, 0x41, 0x4F, 0x67, 0xDC, 0xEA, 0x97, 0xF2, 0xCF, 0xCE, 0xF0, 0xB4, 0xE6, 0x73,
	0x96, 0xAC, 0x74, 0x22, 0xE7, 0xAD, 0x35, 0x85, 0xE2, 0xF9, 0x37, 0xE8, 0x1C, 0x75, 0xDF, 0x6E,
	0x47, 0xF1, 0x1A, 0x71, 0x1D, 0x29, 0xC5, 0x89, 0x6F, 0xB7, 0x62, 0x0E, 0xAA, 0x18, 0xBE, 0x1B,
	0xFC, 0x56, 0x3E, 0x4B, 0xC6, 0xD2, 0x79, 0x20, 0x9A, 0xDB, 0xC0, 0xFE, 0x78, 0xCD, 0x5A, 0xF4,
	0x1F, 0xDD, 0xA8, 0x33, 0x88, 0x07, 0xC7, 0x31, 0xB1, 0x12, 0x10, 0x59, 0x27, 0x80, 0xEC, 0x5F,
	0x60, 0x51, 0x7F, 0xA9, 0x19, 0xB5, 0x4A, 0x0D, 0x2D, 0xE5, 0x7A, 0x9F, 0x93, 0xC9, 0x9C, 0xEF,
	0xA0, 0xE0, 0x3B, 0x4D, 0xAE, 0x2A, 0xF5, 0xB0, 0xC8, 0xEB, 0xBB, 0x3C, 0x83, 0x53, 0x99, 0x61,
	0x17, 0x2B, 0x04, 0x7E, 0xBA, 0x77, 0xD6, 0x26, 0xE1, 0x69, 0x14, 0x63, 0x55, 0x21, 0x0C, 0x7D
};

static const uint8 SB4[256] =
{
	0x30, 0x68, 0x99, 0x1B, 0x87, 0xB9, 0x21, 0x78, 0x50, 0x39, 0xDB, 0xE1, 0x72, 0x09, 0x62, 0x3C,
	0x3E, 0x7E, 0x5E, 0x8E, 0xF1, 0xA0, 0xCC, 0xA3, 0x2A, 0x1D, 0xFB, 0xB6, 0xD6, 0x20, 0xC4, 0x8D,
	0x81, 0x65, 0xF5, 0x89, 0xCB, 0x9D, 0x77, 0xC6, 0x57, 0x43, 0x56, 0x17, 0xD4, 0x40, 0x1A, 0x4D,
	0xC0, 0x63, 0x6C, 0xE3, 0xB7, 0xC8, 0x64, 0x6A, 0x53, 0xAA, 0x38, 0x98, 0x0C, 0xF4, 0x9B, 0xED,
	0x7F, 0x22, 0x76, 0xAF, 0xDD, 0x3A, 0x0B, 0x58, 0x67, 0x88, 0x06, 0xC3, 0x35, 0x0D, 0x01, 0x8B,
	0x8C, 0xC2, 0xE6, 0x5F, 0x02, 0x24, 0x75, 0x93, 0x66, 0x1E, 0xE5, 0xE2, 0x54, 0xD8, 0x10, 0xCE,
	0x7A, 0xE8, 0x08, 0x2C, 0x12, 0x97, 0x32, 0xAB, 0xB4, 0x27, 0x0A, 0x23, 0xDF, 0xEF, 0xCA, 0xD9,
	0xB8, 0xFA, 0xDC, 0x31, 0x6B, 0xD1, 0xAD, 0x19, 0x49, 0xBD, 0x51, 0x96, 0xEE, 0xE4, 0xA8, 0x41,
	0xDA, 0xFF, 0xCD, 0x55, 0x86, 0x36, 0xBE, 0x61, 0x52, 0xF8, 0xBB, 0x0E, 0x82, 0x48, 0x69, 0x9A,
	0xE0, 0x47, 0x9E, 0x5C, 0x04, 0x4B, 0x34, 0x15, 0x79, 0x26, 0xA7, 0xDE, 0x29, 0xAE, 0x92, 0xD7,
	0x84, 0xE9, 0xD2, 0xBA, 0x5D, 0xF3, 0xC5, 0xB0, 0xBF, 0xA4, 0x3B, 0x71, 0x44, 0x46, 0x2B, 0xFC,
	0xEB, 0x6F, 0xD5, 0xF6, 0x14, 0xFE, 0x7C, 0x70, 0x5A, 0x7D, 0xFD, 0x2F, 0x18, 0x83, 0x16, 0xA5,
	0x91, 0x1F, 0x05, 0x95, 0x74, 0xA9, 0xC1, 0x5B, 0x4A, 0x85, 0x6D, 0x13, 0x07, 0x4F, 0x4E, 0x45,
	0xB2, 0x0F, 0xC9, 0x1C, 0xA6, 0xBC, 0xEC, 0x73, 0x90, 0x7B, 0xCF, 0x59, 0x8F, 0xA1, 0xF9, 0x2D,
	0xF2, 0xB1, 0x00, 0x94, 0x37, 0x9F, 0xD0, 0x2E, 0x9C, 0x6E, 0x28, 0x3F, 0x80, 0xF0, 0x3D, 0xD3,
	0x25, 0x8A, 0xB5, 0xE7, 0x42, 0xB3, 0xC7, 0xEA, 0xF7, 0x4C, 0x11, 0x33, 0x03, 0xA2, 0xAC, 0x60
};



uint8 hex2dec(char ch)
{
	if (ch >= '0' && ch <= '9')
		return ch - '0';
	else
		return ch - 'a' + 10;
}

uint8 leftRotate(uint8 n, uint8 d)
{
	return (n << d) | (n >> (8 - d));
}

uint8 rightRotate(uint8 n, uint8 d)
{
	return (n >> d) | (n << (8 - d));
}

uint8* RightShiftBytes(uint8* arr, int arrSize, int amount)//shift the bytes, place them in a new array
{
	uint8* tmp = (uint8*) malloc(amount);
	uint8* newArr = (uint8*)malloc(16 * sizeof(uint8));

	for (int i = 0; i < amount; i++) {
		tmp[i] = arr[arrSize - amount + i];
	}

	for (int i = arrSize - 1; i >= amount; i--) {
		newArr[i] = arr[i - amount];
	}

	for (int i = 0; i < amount; i++) {
		newArr[i] = tmp[i];
	}
	free(tmp);
	return newArr;
}

uint8* LeftShiftBytes(uint8* arr, int arrSize, int amount)//shift the bytes, place them in a new array
{
	uint8* tmp = (uint8*)malloc(amount);
	uint8* newArr = (uint8*)malloc(16 * sizeof(uint8));

	for (int i = 0; i < amount; i++) {
		tmp[i] = arr[i];
	}

	for (int i = 0; i < arrSize - amount; i++) {
		newArr[i] = arr[i + amount];
	}

	for (int i = 0; i < amount; i++) {
		newArr[arrSize - amount + i] = tmp[i];
	}

	free(tmp);
	return newArr;
}

uint8* ShiftArrR(uint8* originalArr, int amount)
{
	int arrSize = 16;
	int byteShiftAmount = amount / 8;
	uint8* arr = RightShiftBytes(originalArr, arrSize, byteShiftAmount);
	amount = amount - byteShiftAmount * 8;

	uint8 carryTmp, carry;
	carry = arr[arrSize - 1] & (0xff >> (8 - amount));//bits that are shifted to byte on right

	for (int i = 0; i < arrSize; i++)
	{
		carryTmp = arr[i] & (0xff >> (8 - amount));//calculate carry for byte on right
		arr[i] >>= amount;//right shift the current byte.
		arr[i] |= rightRotate(carry, amount);//place the bits from coming from byte on left
		carry = carryTmp;
	}

	return arr;
}

uint8* ShiftArrL(uint8* originalArr, int amount)
{
	int arrSize = 16;
	int byteShiftAmount = amount / 8;
	uint8* arr = LeftShiftBytes(originalArr, arrSize, byteShiftAmount);
	amount = amount - byteShiftAmount * 8;

	uint8 carryTmp, carry;
	carry = arr[0] & (0xff << (8 - amount));//bits that are shifted to byte on left

	for (int i = arrSize - 1; i >= 0; i--)
	{
		carryTmp = arr[i] & (0xff << (8 - amount));//calculate carry for byte on left
		arr[i] <<= amount;//left shift the current byte.
		arr[i] |= leftRotate(carry, amount);//place the bits from coming from byte on right
		carry = carryTmp;
	}

	return arr;
}

void XOR_16(uint8* x, uint8* y, uint8* z)
{
	for (int i = 0; i < 16; i++) {
		z[i] = x[i] ^ y[i];
	}
}

void XOR_16wFree(uint8* x, uint8* y, uint8* z)
{
	for (int i = 0; i < 16; i++) {
		z[i] = x[i] ^ y[i];
	}
	free(y);
}

//Substition Layer 1
void SL1(uint8* in, uint8* out)
{
	out[0] = SB1[in[0]];
	out[1] = SB2[in[1]];
	out[2] = SB3[in[2]];
	out[3] = SB4[in[3]];
	out[4] = SB1[in[4]];
	out[5] = SB2[in[5]];
	out[6] = SB3[in[6]];
	out[7] = SB4[in[7]];
	out[8] = SB1[in[8]];
	out[9] = SB2[in[9]];
	out[10] = SB3[in[10]];
	out[11] = SB4[in[11]];
	out[12] = SB1[in[12]];
	out[13] = SB2[in[13]];
	out[14] = SB3[in[14]];
	out[15] = SB4[in[15]];
}

//Substition Layer 2(Inverse of SL1)
void SL2(uint8* in, uint8* out)
{
	out[0] = SB3[in[0]];
	out[1] = SB4[in[1]];
	out[2] = SB1[in[2]];
	out[3] = SB2[in[3]];
	out[4] = SB3[in[4]];
	out[5] = SB4[in[5]];
	out[6] = SB1[in[6]];
	out[7] = SB2[in[7]];
	out[8] = SB3[in[8]];
	out[9] = SB4[in[9]];
	out[10] = SB1[in[10]];
	out[11] = SB2[in[11]];
	out[12] = SB3[in[12]];
	out[13] = SB4[in[13]];
	out[14] = SB1[in[14]];
	out[15] = SB2[in[15]];
}

//Diffusion layer
void A(uint8* in, uint8* out)
{
	out[0] = in[3] ^ in[4] ^ in[6] ^ in[8] ^ in[9] ^ in[13] ^ in[14];
	out[1] = in[2] ^ in[5] ^ in[7] ^ in[8] ^ in[9] ^ in[12] ^ in[15];
	out[2] = in[1] ^ in[4] ^ in[6] ^ in[10] ^ in[11] ^ in[12] ^ in[15];
	out[3] = in[0] ^ in[5] ^ in[7] ^ in[10] ^ in[11] ^ in[13] ^ in[14];
	out[4] = in[0] ^ in[2] ^ in[5] ^ in[8] ^ in[11] ^ in[14] ^ in[15];
	out[5] = in[1] ^ in[3] ^ in[4] ^ in[9] ^ in[10] ^ in[14] ^ in[15];
	out[6] = in[0] ^ in[2] ^ in[7] ^ in[9] ^ in[10] ^ in[12] ^ in[13];
	out[7] = in[1] ^ in[3] ^ in[6] ^ in[8] ^ in[11] ^ in[12] ^ in[13];
	out[8] = in[0] ^ in[1] ^ in[4] ^ in[7] ^ in[10] ^ in[13] ^ in[15];
	out[9] = in[0] ^ in[1] ^ in[5] ^ in[6] ^ in[11] ^ in[12] ^ in[14];
	out[10] = in[2] ^ in[3] ^ in[5] ^ in[6] ^ in[8] ^ in[13] ^ in[15];
	out[11] = in[2] ^ in[3] ^ in[4] ^ in[7] ^ in[9] ^ in[12] ^ in[14];
	out[12] = in[1] ^ in[2] ^ in[6] ^ in[7] ^ in[9] ^ in[11] ^ in[12];
	out[13] = in[0] ^ in[3] ^ in[6] ^ in[7] ^ in[8] ^ in[10] ^ in[13];
	out[14] = in[0] ^ in[3] ^ in[4] ^ in[5] ^ in[9] ^ in[11] ^ in[14];
	out[15] = in[1] ^ in[2] ^ in[4] ^ in[5] ^ in[8] ^ in[10] ^ in[15];
}

/*Round Functions(F0,FE) takes 16 bytes of plaintext
and generates an intermediate val of 16bytes
*/

//Odd Round Function
void F0(uint8* D, uint8* RK, uint8* out)
{
	//res1, res2 are auxillary arrays for storing the results of XOR_16 and SL1
	uint8 res1[16];
	uint8 res2[16];
	XOR_16(D, RK, res1);
	SL1(res1, res2);
	A(res2, out);
}

//Even Round Function
void FE(uint8* D, uint8* RK, uint8* out)
{
	//res1, res2 are auxillary arrays for storing the results of XOR_16 and SL1
	uint8 res1[16];
	uint8 res2[16];
	XOR_16(D, RK, res1);
	SL2(res1, res2);
	A(res2, out);
}

void GenerateRoundKeys(uint8* W0, uint8* W1, uint8* W2, uint8* W3)
{
	//Producing encryption round keys
	//Producing encryption round keys can be parallelized.
	//However since we do this once for all blocks, it is faster to compute in CPU.
	//ShiftArr functions return array from heap, must free.
	XOR_16wFree(W0, ShiftArrR(W1, 19), &ek[0]);
	XOR_16wFree(W1, ShiftArrR(W2, 19), &ek[16]);
	XOR_16wFree(W2, ShiftArrR(W3, 19), &ek[32]);
	XOR_16wFree(W3, ShiftArrR(W0, 19), &ek[48]);
	XOR_16wFree(W0, ShiftArrR(W1, 31), &ek[64]);
	XOR_16wFree(W1, ShiftArrR(W2, 31), &ek[80]);
	XOR_16wFree(W2, ShiftArrR(W3, 31), &ek[96]);
	XOR_16wFree(W3, ShiftArrR(W0, 31), &ek[112]);
	XOR_16wFree(W0, ShiftArrL(W1, 61), &ek[128]);
	XOR_16wFree(W1, ShiftArrL(W2, 61), &ek[144]);
	XOR_16wFree(W2, ShiftArrL(W3, 61), &ek[160]);
	XOR_16wFree(W3, ShiftArrL(W0, 61), &ek[176]);
	XOR_16wFree(W0, ShiftArrL(W1, 31), &ek[192]);
	XOR_16wFree(W1, ShiftArrL(W2, 31), &ek[208]);
	XOR_16wFree(W2, ShiftArrL(W3, 31), &ek[224]);
	XOR_16wFree(W3, ShiftArrL(W0, 31), &ek[240]);
	XOR_16wFree(W0, ShiftArrL(W1, 19), &ek[256]);
}


void GenerateDecRoundKeys(uint8 numOfRounds)
{
	int N = numOfRounds - 1;
	int k = 1;

	for (int i = 0; i < 16; i++)
	{
		dk[i] = ek[16 * N + i];
	}


	for (int i = N - 1; i >= 1; i--)
	{
		A(&ek[i * 16], &dk[k * 16]);
		k++;
	}

	for (int i = 0; i < 16; i++)
	{
		dk[k * 16 + i] = ek[i];
	}
}

//Odd Round Function
__device__ void F0_d(uint8* D, const uint8* RK, uint8* SB1, uint8* SB2, uint8* SB3, uint8* SB4)
{
	uint8 aux[16];//auxilary array for keeping the results of Diffusion layer

  	//XOR with the round key
	#pragma unroll
	for (int i = 0; i < 16; i++) {
		D[i] = D[i] ^ RK[i];
	}
	//Substition Layer(SL1)
	D[0] = SB1[D[0]];
	D[1] = SB2[D[1]];
	D[2] = SB3[D[2]];
	D[3] = SB4[D[3]];
	D[4] = SB1[D[4]];
	D[5] = SB2[D[5]];
	D[6] = SB3[D[6]];
	D[7] = SB4[D[7]];
	D[8] = SB1[D[8]];
	D[9] = SB2[D[9]];
	D[10] = SB3[D[10]];
	D[11] = SB4[D[11]];
	D[12] = SB1[D[12]];
	D[13] = SB2[D[13]];
	D[14] = SB3[D[14]];
	D[15] = SB4[D[15]];
	//Diffusion layer
	aux[0] = D[3] ^ D[4] ^ D[6] ^ D[8] ^ D[9] ^ D[13] ^ D[14];
	aux[1] = D[2] ^ D[5] ^ D[7] ^ D[8] ^ D[9] ^ D[12] ^ D[15];
	aux[2] = D[1] ^ D[4] ^ D[6] ^ D[10] ^ D[11] ^ D[12] ^ D[15];
	aux[3] = D[0] ^ D[5] ^ D[7] ^ D[10] ^ D[11] ^ D[13] ^ D[14];
	aux[4] = D[0] ^ D[2] ^ D[5] ^ D[8] ^ D[11] ^ D[14] ^ D[15];
	aux[5] = D[1] ^ D[3] ^ D[4] ^ D[9] ^ D[10] ^ D[14] ^ D[15];
	aux[6] = D[0] ^ D[2] ^ D[7] ^ D[9] ^ D[10] ^ D[12] ^ D[13];
	aux[7] = D[1] ^ D[3] ^ D[6] ^ D[8] ^ D[11] ^ D[12] ^ D[13];
	aux[8] = D[0] ^ D[1] ^ D[4] ^ D[7] ^ D[10] ^ D[13] ^ D[15];
	aux[9] = D[0] ^ D[1] ^ D[5] ^ D[6] ^ D[11] ^ D[12] ^ D[14];
	aux[10] = D[2] ^ D[3] ^ D[5] ^ D[6] ^ D[8] ^ D[13] ^ D[15];
	aux[11] = D[2] ^ D[3] ^ D[4] ^ D[7] ^ D[9] ^ D[12] ^ D[14];
	aux[12] = D[1] ^ D[2] ^ D[6] ^ D[7] ^ D[9] ^ D[11] ^ D[12];
	aux[13] = D[0] ^ D[3] ^ D[6] ^ D[7] ^ D[8] ^ D[10] ^ D[13];
	aux[14] = D[0] ^ D[3] ^ D[4] ^ D[5] ^ D[9] ^ D[11] ^ D[14];
	aux[15] = D[1] ^ D[2] ^ D[4] ^ D[5] ^ D[8] ^ D[10] ^ D[15];

	//put the result into plaintext registers
	#pragma unroll
	for (int i = 0; i < 16; i++) {
		D[i] = aux[i];
	}
}

//Even Round Function
__device__ void FE_d(uint8* D, const uint8* RK, uint8* SB1, uint8* SB2, uint8* SB3, uint8* SB4)
{
	uint8 aux[16];//auxilary array for keeping the results of Diffusion layer

  	//XOR with the round key
	#pragma unroll
	for (int i = 0; i < 16; i++) {
		D[i] = D[i] ^ RK[i];
	}
	//Substition Layer(SL2)
	D[0] = SB3[D[0]];
	D[1] = SB4[D[1]];
	D[2] = SB1[D[2]];
	D[3] = SB2[D[3]];
	D[4] = SB3[D[4]];
	D[5] = SB4[D[5]];
	D[6] = SB1[D[6]];
	D[7] = SB2[D[7]];
	D[8] = SB3[D[8]];
	D[9] = SB4[D[9]];
	D[10] = SB1[D[10]];
	D[11] = SB2[D[11]];
	D[12] = SB3[D[12]];
	D[13] = SB4[D[13]];
	D[14] = SB1[D[14]];
	D[15] = SB2[D[15]];
	//Diffusion layer
	aux[0] = D[3] ^ D[4] ^ D[6] ^ D[8] ^ D[9] ^ D[13] ^ D[14];
	aux[1] = D[2] ^ D[5] ^ D[7] ^ D[8] ^ D[9] ^ D[12] ^ D[15];
	aux[2] = D[1] ^ D[4] ^ D[6] ^ D[10] ^ D[11] ^ D[12] ^ D[15];
	aux[3] = D[0] ^ D[5] ^ D[7] ^ D[10] ^ D[11] ^ D[13] ^ D[14];
	aux[4] = D[0] ^ D[2] ^ D[5] ^ D[8] ^ D[11] ^ D[14] ^ D[15];
	aux[5] = D[1] ^ D[3] ^ D[4] ^ D[9] ^ D[10] ^ D[14] ^ D[15];
	aux[6] = D[0] ^ D[2] ^ D[7] ^ D[9] ^ D[10] ^ D[12] ^ D[13];
	aux[7] = D[1] ^ D[3] ^ D[6] ^ D[8] ^ D[11] ^ D[12] ^ D[13];
	aux[8] = D[0] ^ D[1] ^ D[4] ^ D[7] ^ D[10] ^ D[13] ^ D[15];
	aux[9] = D[0] ^ D[1] ^ D[5] ^ D[6] ^ D[11] ^ D[12] ^ D[14];
	aux[10] = D[2] ^ D[3] ^ D[5] ^ D[6] ^ D[8] ^ D[13] ^ D[15];
	aux[11] = D[2] ^ D[3] ^ D[4] ^ D[7] ^ D[9] ^ D[12] ^ D[14];
	aux[12] = D[1] ^ D[2] ^ D[6] ^ D[7] ^ D[9] ^ D[11] ^ D[12];
	aux[13] = D[0] ^ D[3] ^ D[6] ^ D[7] ^ D[8] ^ D[10] ^ D[13];
	aux[14] = D[0] ^ D[3] ^ D[4] ^ D[5] ^ D[9] ^ D[11] ^ D[14];
	aux[15] = D[1] ^ D[2] ^ D[4] ^ D[5] ^ D[8] ^ D[10] ^ D[15];

	//put the result into plaintext registers
	#pragma unroll
	for (int i = 0; i < 16; i++) {
		D[i] = aux[i];
	}
}



template <unsigned int keySize>
__global__ void Decrypt(uint8* cipherText, unsigned long textSize, uint8* dk, uint8* SB_gmem, uint8* IV)
{
	unsigned int tid = threadIdx.x;
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	uint8 cipherTextR_1[16];//registers keeping the cipherText.
	uint8 cipherTextR_2[16];//registers keeping the cipherText.

	__shared__ uint8 keySmem[272];//each round key is 16 bytes, there are 17 round keys 272 bytes
	__shared__ uint8 SB1[256];
	__shared__ uint8 SB2[256];
	__shared__ uint8 SB3[256];
	__shared__ uint8 SB4[256];

	//Load decryption round keys to shared memory.
	keySmem[tid] = dk[tid];
	//rest of the bytes(272 bytes) are loaded by first 16 threads.
	if (tid < 16) {
		keySmem[256 + tid] = dk[256 + tid];
	}

	//Load SB tables to shared memory.(1024 bytes loaded by 256 threads)
	SB1[tid] = SB_gmem[tid];
	SB2[tid] = SB_gmem[tid + 256];
	SB3[tid] = SB_gmem[tid + 512];
	SB4[tid] = SB_gmem[tid + 768];

	//Load the plaintext to registers
	//Each thread is responsible for 16 bytes.

	if(idx == 0)//first thread is different than other threads.Uses IV.
	{
		for (int i = 0; i < 16; i++)
		{
			cipherTextR_1[i] = IV[i];
		}
	}
	else
	{
		for (int i = 0; i < 16; i++)
		{
			cipherTextR_1[i] = cipherText[16 * idx -16 + i];//non-coalasced access, slow
		}
	}

	for (int i = 0; i < 16; i++)
	{
		cipherTextR_2[i] = cipherText[16 * idx + i];//non-coalasced access, slow
	}

	__syncthreads();

	if (keySize == 16)//128-bit keys
	{
		F0_d(cipherTextR_2, &keySmem[0], SB1, SB2, SB3, SB4);//ek1...   
		FE_d(cipherTextR_2, &keySmem[16], SB1, SB2, SB3, SB4);
		F0_d(cipherTextR_2, &keySmem[32], SB1, SB2, SB3, SB4);
		FE_d(cipherTextR_2, &keySmem[48], SB1, SB2, SB3, SB4);
		F0_d(cipherTextR_2, &keySmem[64], SB1, SB2, SB3, SB4);
		FE_d(cipherTextR_2, &keySmem[80], SB1, SB2, SB3, SB4);
		F0_d(cipherTextR_2, &keySmem[96], SB1, SB2, SB3, SB4);
		FE_d(cipherTextR_2, &keySmem[112], SB1, SB2, SB3, SB4);
		F0_d(cipherTextR_2, &keySmem[128], SB1, SB2, SB3, SB4);
		FE_d(cipherTextR_2, &keySmem[144], SB1, SB2, SB3, SB4);
		F0_d(cipherTextR_2, &keySmem[160], SB1, SB2, SB3, SB4);//...ek11

		#pragma unroll
		for (int i = 0; i < 16; i++) {
			cipherTextR_2[i] = cipherTextR_2[i] ^ keySmem[176 + i];//ek12
		}

		cipherTextR_2[0] = SB3[cipherTextR_2[0]];
		cipherTextR_2[1] = SB4[cipherTextR_2[1]];
		cipherTextR_2[2] = SB1[cipherTextR_2[2]];
		cipherTextR_2[3] = SB2[cipherTextR_2[3]];
		cipherTextR_2[4] = SB3[cipherTextR_2[4]];
		cipherTextR_2[5] = SB4[cipherTextR_2[5]];
		cipherTextR_2[6] = SB1[cipherTextR_2[6]];
		cipherTextR_2[7] = SB2[cipherTextR_2[7]];
		cipherTextR_2[8] = SB3[cipherTextR_2[8]];
		cipherTextR_2[9] = SB4[cipherTextR_2[9]];
		cipherTextR_2[10] = SB1[cipherTextR_2[10]];
		cipherTextR_2[11] = SB2[cipherTextR_2[11]];
		cipherTextR_2[12] = SB3[cipherTextR_2[12]];
		cipherTextR_2[13] = SB4[cipherTextR_2[13]];
		cipherTextR_2[14] = SB1[cipherTextR_2[14]];
		cipherTextR_2[15] = SB2[cipherTextR_2[15]];

		#pragma unroll
		for (int i = 0; i < 16; i++) {
			cipherTextR_2[i] = cipherTextR_2[i] ^ keySmem[192 + i];//ek13
		}

		//XOR with the previous block.
		#pragma unroll
		for (int i = 0; i < 16; i++) {
			cipherTextR_2[i] = cipherTextR_2[i] ^ cipherTextR_1[i];
		}

		//Write back to global memory
		for (int i = 0; i < 16; i++)
		{
			cipherText[16 * idx + i] = cipherTextR_2[i];
		}
	}
	else if (keySize == 24)//192-bit keys
	{
		F0_d(cipherTextR_2, &keySmem[0], SB1, SB2, SB3, SB4);//ek1...   
		FE_d(cipherTextR_2, &keySmem[16], SB1, SB2, SB3, SB4);
		F0_d(cipherTextR_2, &keySmem[32], SB1, SB2, SB3, SB4);
		FE_d(cipherTextR_2, &keySmem[48], SB1, SB2, SB3, SB4);
		F0_d(cipherTextR_2, &keySmem[64], SB1, SB2, SB3, SB4);
		FE_d(cipherTextR_2, &keySmem[80], SB1, SB2, SB3, SB4);
		F0_d(cipherTextR_2, &keySmem[96], SB1, SB2, SB3, SB4);
		FE_d(cipherTextR_2, &keySmem[112], SB1, SB2, SB3, SB4);
		F0_d(cipherTextR_2, &keySmem[128], SB1, SB2, SB3, SB4);
		FE_d(cipherTextR_2, &keySmem[144], SB1, SB2, SB3, SB4);
		F0_d(cipherTextR_2, &keySmem[160], SB1, SB2, SB3, SB4);
		FE_d(cipherTextR_2, &keySmem[176], SB1, SB2, SB3, SB4);
		F0_d(cipherTextR_2, &keySmem[192], SB1, SB2, SB3, SB4);//ek13

		#pragma unroll
		for (int i = 0; i < 16; i++) {
			cipherTextR_2[i] = cipherTextR_2[i] ^ keySmem[208 + i];//ek14
		}

		cipherTextR_2[0] = SB3[cipherTextR_2[0]];
		cipherTextR_2[1] = SB4[cipherTextR_2[1]];
		cipherTextR_2[2] = SB1[cipherTextR_2[2]];
		cipherTextR_2[3] = SB2[cipherTextR_2[3]];
		cipherTextR_2[4] = SB3[cipherTextR_2[4]];
		cipherTextR_2[5] = SB4[cipherTextR_2[5]];
		cipherTextR_2[6] = SB1[cipherTextR_2[6]];
		cipherTextR_2[7] = SB2[cipherTextR_2[7]];
		cipherTextR_2[8] = SB3[cipherTextR_2[8]];
		cipherTextR_2[9] = SB4[cipherTextR_2[9]];
		cipherTextR_2[10] = SB1[cipherTextR_2[10]];
		cipherTextR_2[11] = SB2[cipherTextR_2[11]];
		cipherTextR_2[12] = SB3[cipherTextR_2[12]];
		cipherTextR_2[13] = SB4[cipherTextR_2[13]];
		cipherTextR_2[14] = SB1[cipherTextR_2[14]];
		cipherTextR_2[15] = SB2[cipherTextR_2[15]];

		#pragma unroll
		for (int i = 0; i < 16; i++) {
			cipherTextR_2[i] = cipherTextR_2[i] ^ keySmem[224 + i];//ek15
		}

		//XOR with the previous block.
		#pragma unroll
		for (int i = 0; i < 16; i++) {
			cipherTextR_2[i] = cipherTextR_2[i] ^ cipherTextR_1[i];
		}

		//Write back to global memory
		for (int i = 0; i < 16; i++)
		{
			cipherText[16 * idx + i] = cipherTextR_2[i];
		}

	}
	else//256-bit keys
	{
		F0_d(cipherTextR_2, &keySmem[0], SB1, SB2, SB3, SB4);//ek1...    
		FE_d(cipherTextR_2, &keySmem[16], SB1, SB2, SB3, SB4);
		F0_d(cipherTextR_2, &keySmem[32], SB1, SB2, SB3, SB4);
		FE_d(cipherTextR_2, &keySmem[48], SB1, SB2, SB3, SB4);
		F0_d(cipherTextR_2, &keySmem[64], SB1, SB2, SB3, SB4);
		FE_d(cipherTextR_2, &keySmem[80], SB1, SB2, SB3, SB4);
		F0_d(cipherTextR_2, &keySmem[96], SB1, SB2, SB3, SB4);
		FE_d(cipherTextR_2, &keySmem[112], SB1, SB2, SB3, SB4);
		F0_d(cipherTextR_2, &keySmem[128], SB1, SB2, SB3, SB4);
		FE_d(cipherTextR_2, &keySmem[144], SB1, SB2, SB3, SB4);
		F0_d(cipherTextR_2, &keySmem[160], SB1, SB2, SB3, SB4);
		FE_d(cipherTextR_2, &keySmem[176], SB1, SB2, SB3, SB4);
		F0_d(cipherTextR_2, &keySmem[192], SB1, SB2, SB3, SB4);
		FE_d(cipherTextR_2, &keySmem[208], SB1, SB2, SB3, SB4);
		F0_d(cipherTextR_2, &keySmem[224], SB1, SB2, SB3, SB4);//ek15

		#pragma unroll
		for (int i = 0; i < 16; i++) {
			cipherTextR_2[i] = cipherTextR_2[i] ^ keySmem[240 + i];//ek16
		}

		cipherTextR_2[0] = SB3[cipherTextR_2[0]];
		cipherTextR_2[1] = SB4[cipherTextR_2[1]];
		cipherTextR_2[2] = SB1[cipherTextR_2[2]];
		cipherTextR_2[3] = SB2[cipherTextR_2[3]];
		cipherTextR_2[4] = SB3[cipherTextR_2[4]];
		cipherTextR_2[5] = SB4[cipherTextR_2[5]];
		cipherTextR_2[6] = SB1[cipherTextR_2[6]];
		cipherTextR_2[7] = SB2[cipherTextR_2[7]];
		cipherTextR_2[8] = SB3[cipherTextR_2[8]];
		cipherTextR_2[9] = SB4[cipherTextR_2[9]];
		cipherTextR_2[10] = SB1[cipherTextR_2[10]];
		cipherTextR_2[11] = SB2[cipherTextR_2[11]];
		cipherTextR_2[12] = SB3[cipherTextR_2[12]];
		cipherTextR_2[13] = SB4[cipherTextR_2[13]];
		cipherTextR_2[14] = SB1[cipherTextR_2[14]];
		cipherTextR_2[15] = SB2[cipherTextR_2[15]];

		#pragma unroll
		for (int i = 0; i < 16; i++) {
			cipherTextR_2[i] = cipherTextR_2[i] ^ keySmem[256 + i];//ek17
		}

		//XOR with the previous block.
		#pragma unroll
		for (int i = 0; i < 16; i++) {
			cipherTextR_2[i] = cipherTextR_2[i] ^ cipherTextR_1[i];
		}

		//Write back to global memory
		for (int i = 0; i < 16; i++)
		{
			cipherText[16 * idx + i] = cipherTextR_2[i];
		}
	}

}


int main(void)
{
	/////////INPUT PART BEGIN//////////////////////

	//Device pointers:
	uint8* deviceArr, *dk_d, *SB_dev, *IV_d;

	FILE *file;
	uint8* inputText;//ciphertext input
	unsigned long int fileLen, textSize;
	uint8 numOfRounds;
    const uint8 keySize = 32;
    uint8 key[32] = {0x00, 0x01, 0x02, 0x03, 0x04, 0x05, 0x06, 0x07, 0x08, 0x09, 0x0a, 0x0b, 0x0c, 0x0d, 0x0e, 0x0f,
					0x10, 0x11, 0x12, 0x13, 0x14, 0x15, 0x16, 0x17, 0x18, 0x19, 0x1a, 0x1b, 0x1c, 0x1d, 0x1e, 0x1f};

	uint8 IV[16] = {0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00};

	file = fopen("../input.txt", "r");
	if (file)
	{
		char buf[2];

		fseek(file, 0, SEEK_END);
		fileLen = ftell(file);
		fseek(file, 0, SEEK_SET);
		textSize = fileLen / 2;
		inputText = (uint8*)malloc(textSize);

		for (int i = 0; i < textSize; i++)
		{
			buf[0] = fgetc(file);
			buf[1] = fgetc(file);
			uint8 hexVal = (uint8)strtol(buf, NULL, 16);
			inputText[i] = hexVal;
		}
	}
	else
	{
		printf("File not found.\n");
		return -1;
	}
	/////////INPUT PART END//////////////////////

	if (keySize == 16)
		numOfRounds = 13;
	else if (keySize == 24)
		numOfRounds = 15;
	else
		numOfRounds = 17;


	uint8 KL[16];//KL = leftmost 16 bytes of key 
	uint8 KR[16];//KR = rightmost 16 bytes of key

	/*
	Most significant byte is stored in 0th index.
	KL = leftmost 16 bytes of key
	KR = rightmost 16 bytes of key
	*/

	for (int i = 0; i < 16; i++)
	{
		KL[i] = key[i];
	}

	for (int i = 0; i < 16; i++)
	{
		KR[i] = key[i + 16];
	}

	uint8* CK1, *CK2, *CK3;
	if (keySize == 16) {
		CK1 = C1;
		CK2 = C2;
		CK3 = C3;
	}
	else if (keySize == 24) {
		CK1 = C2;
		CK2 = C3;
		CK3 = C1;
	}
	else {
		CK1 = C3;
		CK2 = C1;
		CK3 = C2;
	}

	//Calculate round key generators W0,W1,W2,W3
	uint8* W0 = KL;
	uint8 W1[16];
	uint8 W2[16];
	uint8 W3[16];
	uint8 Fres[16];//auxilary array

	/*
	W0, W1, W2, W3 are calculated only once and used for all blocks.
	Since the key data W0 and CK1 are small enough this key generators are calculated in CPU.
	W1 needed for calc of W2, W2 needed for calc of W3.
	F0 and FE are also used in the encryption process.
	*/

	F0(W0, CK1, Fres);
	XOR_16(Fres, KR, W1);

	FE(W1, CK2, Fres);
	XOR_16(Fres, W0, W2);

	F0(W2, CK3, Fres);
	XOR_16(Fres, W1, W3);

	GenerateRoundKeys(W0, W1, W2, W3);

	/*
	Because each thread will process 16 bytes we need textSize/16 threads in total.
	Then thread number per block is: ceil(textSize/(16*blockSize)) bytes.
	To decide blockSize we must consider the main occupancy limiter, in this case number of registers per SM.
	Based on NVIDIA's programming guide Number of 32-bit registers per multiprocessor for compute capability >= 5.0 is 64K.
	In this code 16 registers used for plaintext, 16 registers auxilary, +1 by itself, each thread uses 33 registers.
	Then blocksize must be smaller than 64k/33. And larger than 272 since first 272 threads loads the shared memory.
	512, 1024 are available blockSizes.
	256 can also be tried but number of threads loading the shared memory must be decreased.
	Keeping the round keys in registers results in low number of warps per SM therefore poor performance.
	*/

	int blockSize = 256;
	int numOfBlocks = ceil((float)(textSize) / (16 * blockSize));


	//Decryption round keys are derived from the encryption round keys which is generated by GenerateRoundKeys.
	GenerateDecRoundKeys(numOfRounds);

	uint8* resPlainText = (uint8*)malloc(textSize);
	hipMalloc((void**)& deviceArr, textSize);
	hipMalloc((void**)& dk_d, 272);
	hipMalloc((void**)& IV_d, 16);
	hipMalloc((void**)& SB_dev, 1024);


	//START TIMER.
	using namespace std::chrono;
	high_resolution_clock::time_point start = high_resolution_clock::now();

	hipMemcpy(deviceArr, inputText, textSize, hipMemcpyHostToDevice);
	hipMemcpy(dk_d, dk, 272, hipMemcpyHostToDevice);
	hipMemcpy(IV_d, IV, 16, hipMemcpyHostToDevice);
	//Move Substition layer tables to global memory.(will be moved to shared memory in the kernel.)
	hipMemcpy(SB_dev, SB1, 256, hipMemcpyHostToDevice);
	hipMemcpy(SB_dev + 256, SB2, 256, hipMemcpyHostToDevice);
	hipMemcpy(SB_dev + 512, SB3, 256, hipMemcpyHostToDevice);
	hipMemcpy(SB_dev + 768, SB4, 256, hipMemcpyHostToDevice);
	Decrypt<keySize> << <numOfBlocks, blockSize >> >  (deviceArr, textSize, dk_d, SB_dev ,IV_d);
	hipMemcpy(resPlainText, deviceArr, textSize, hipMemcpyDeviceToHost);

	//END TIMER; PRINT ELAPSED TIME.
	high_resolution_clock::time_point end = high_resolution_clock::now();
	duration<double> timeElapsed = duration_cast<duration<double>>(end - start);
	std::cout << "Time elapsed: " << timeElapsed.count() << std::endl;		


	//Print/write to file
	FILE *f = fopen("output.txt", "w");
	for (int i = 0; i < textSize; i++) {
		fprintf(f, "%02x", resPlainText[i]);
	}
	fclose(f);

	//free
	hipFree(deviceArr);
	hipFree(dk_d);
	free(resPlainText);



	return 0;
}